#include <memory>
#include <kernels.cu>
using namespace std;

#define MAX_THRpBLK2D 32

enum Dir { H2D= hipMemcpyHostToDevice, D2H= hipMemcpyDeviceToHost };

template<class T>
struct DPtr{
  DPtr(int N) { hipMalloc(&data_, N*N*sizeof(T)); }
  ~DPtr() { hipFree(data_); }
  void copy(T* a, int N, Dir dir) {
    if(dir == Dir::H2D) hipMemcpy(data_, a, sizeof(T)*N, dir);
    else hipMemcpy(a, data_, sizeof(T)*N, dir);
  }
  T* get() { return data_; }
private:
  T* data_;
}

int main(){
  int N;
  scanf("%d\n", &N);
  unique_ptr<int[]> g(new int[N*N]);
  DPtr<int> d_g(N*N);
  for(int i=0; i<N; i++)
    for(int j=0; j<N; j++)
      scanf("%d", &g[i*N+j]);

  // simple GPU Floyd-Warshall
  d_g.copy(g.get(), N*N, Dir::H2D);
  dim3 bs(MAX_THRpBLK2D, MAX_THRpBLK2D);
  dim3 gs(N/bs.x, N/bs.y);
  for(int k=0; k<N; k++) fw<<<gs,bs>>>(d_g.get(), N, k);
  unique_ptr<int[]> result_simple(new int[N*N]);
  d_g.copy(result_simple.get(), N*N, Dir::D2H);

  // block algo
  d_g.copy(g.get(), N*N, Dir::H2D);
  const int n= MAX_THRpBLK2D, B= N/n;
  for(int b=0; b< B; b++){
    phase1<<<1,bs>>>(d_g, b*n);
    phase2<<<(B-1,2),bs>>>(d_g, b*n, b);
    phase3<<<(B-1,B-1),bs>>>(d_g, b*n, b);
  }
  unique_ptr<int[]> result_block(new int[N*N]);
  d_g.copy(result_block.get(), N*N, Dir::D2H);

  for(int i=0; i<N; i++)
    for(int j=0; j<N; j++)
      if(result_simple[i*N+j] != result_block[i*N+j]){
        printf("[check]: Error at (%d,%d)! simple/block =\n\t%d\n\t%d\n", i,j, result_simple[i*N+j],
            result_block[i*N+j]);
        exit(1);
      }

  return 0;
}

void (){
  for(int i=0; i<d; i++){
    phase1Wrp();
    phase2Wrp();
    phase3Wrp();
  }
}
