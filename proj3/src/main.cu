#include <cstdio>
#include <memory>
#include <chrono>
#include <hip/hip_runtime.h>
#include "kernels.cu"
using namespace std;

#define gpuErrchk(ans) gpuAssert((ans), __FILE__, __LINE__)
#define MAX_THRpBLK2D 32

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

enum Dir { H2D, D2H };

template<class T>
struct DPtr{
  DPtr(int N) { gpuErrchk(hipMalloc(&data_, N*sizeof(T))); }
  ~DPtr() { hipFree(data_); }
  void copy(T* a, int N, Dir dir) {
    if(dir == Dir::H2D) gpuErrchk(hipMemcpy(data_, a, sizeof(T)*N, hipMemcpyHostToDevice));
    else gpuErrchk(hipMemcpy(a, data_, sizeof(T)*N, hipMemcpyDeviceToHost));
  }
  T* get() { return data_; }
  operator T*() { return data_; }
private:
  T* data_;
};

void printG(unique_ptr<int[]>& g, int N){
  for(int i=0; i<N; i++){
    for(int j=0; j<N; j++)
      printf("%3d\t", g[i*N+j]);
    printf("\n");
  }
  printf("_____________________________________\n");
}

typedef chrono::duration<float, ratio<1>> Duration_fsec;

bool test(const unique_ptr<int[]>& toCheck, const unique_ptr<int[]>& truth, const int N, string name){
  for(int i=0; i<N; i++)
    for(int j=0; j<N; j++)
      if(toCheck[i*N+j] != truth[i*N+j]){
        printf("[test/%s]: Error at (%d,%d)! toCheck/truth =\n\t%d\n\t%d\n", name.c_str(), i,j, toCheck[i*N+j],
            truth[i*N+j]);
        return false;
      }
  return true;
}

// CPU Floyd-Warshall
Duration_fsec run_cpu(const unique_ptr<int[]>& g, const int N, unique_ptr<int[]>& result_cpu){
  for(int i=0; i<N*N; i++) result_cpu[i]= g[i];     // Work on a copy of the data
  /*clk*/auto start= chrono::system_clock::now();
  for(int k=0; k<N; k++)
    for(int i=0; i<N; i++)
      for(int j=0; j<N; j++)
        if(result_cpu[i*N+j] > result_cpu[i*N+k]+result_cpu[k*N+j])
          result_cpu[i*N+j]= result_cpu[i*N+k]+result_cpu[k*N+j];
  /*clk*/auto cpu_time= chrono::duration_cast<Duration_fsec>(chrono::system_clock::now() - start);
  printf("CPU calc done: %1.3fs\n", cpu_time.count());
#ifdef LOG
  fprintf(logfile, "%1.3f;", cpu_time.count());
#endif
  return cpu_time;
}

// simple GPU Floyd-Warshall
Duration_fsec run_GPUsimple(const unique_ptr<int[]>&g, const int N, const unique_ptr<int[]>& groundTruth){
  DPtr<int> d_g(N*N);
  unique_ptr<int[]> result_simple(new int[N*N]);
  dim3 bs(MAX_THRpBLK2D, MAX_THRpBLK2D);
  if(N<MAX_THRpBLK2D) bs= dim3(N,N);
  dim3 gs(N/bs.x, N/bs.y);
  printf("Launching GPU simple algo...\n");
  /*clk*/auto start= chrono::system_clock::now();
  d_g.copy(g.get(), N*N, Dir::H2D);
  for(int k=0; k<N; k++) fw<<<gs,bs>>>(d_g, N, k);
  d_g.copy(result_simple.get(), N*N, Dir::D2H);
  /*clk*/auto GPUSimple_time= chrono::duration_cast<Duration_fsec>(chrono::system_clock::now() - start);
  printf("GPU simple kernel done: %.3f\n", GPUSimple_time.count());
#ifdef LOG
  fprintf(logfile, "%1.3f;", GPUSimple_time.count());
#endif
  auto check= test(result_simple, groundTruth, N, "GPUsimple");
  if(!check){
    printf("[GPUsimple]: Test FAILED!\n");
    exit(1);
  }
  return GPUSimple_time;
}

// GPU block algo
Duration_fsec run_GPUblock(const unique_ptr<int[]>&g, const int N, const unique_ptr<int[]>& groundTruth ){
  DPtr<int> d_g(N*N);
  unique_ptr<int[]> result_block(new int[N*N]);
  constexpr const int n= MAX_THRpBLK2D;
  const int B= N/n;
  dim3 bs(MAX_THRpBLK2D, MAX_THRpBLK2D);
  if(N<MAX_THRpBLK2D) bs= dim3(N,N);
  dim3 gs(N/bs.x, N/bs.y);

  printf("Launching GPU block algo with %d primary blocks\n", B);
  /*clk*/auto start= chrono::system_clock::now();
  d_g.copy(g.get(), N*N, Dir::H2D);
  for(int b=0; b<B; b++){
    phase1<n> <<<1,bs>>>(d_g, b*n, N);
    phase2<n> <<<dim3(B-1,2),bs>>>(d_g, b*n, b, N);
    phase3<n> <<<dim3(B-1,B-1),bs>>>(d_g, b*n, b, N);
  }
  d_g.copy(result_block.get(), N*N, Dir::D2H);
  /*clk*/auto GPUBlock_time= chrono::duration_cast<Duration_fsec>(chrono::system_clock::now() - start);
  printf("GPU block kernel done: %.3f\n", GPUBlock_time.count());
#ifdef LOG
  fprintf(logfile, "%.3f;", GPUBlock_time.count());
#endif
  auto check= test(result_block, groundTruth, N, "GPUblock");
  if(!check){
    printf("[GPUblock]: Test FAILED!\n");
    exit(1);
  }
  return GPUBlock_time; 
}

int main(){
  int N;
  scanf("%d\n", &N);
  unique_ptr<int[]> g(new int[N*N]), groundTruth(new int[N*N]);
  for(int i=0; i<N; i++)
    for(int j=0; j<N; j++)
      scanf("%d", &g[i*N+j]);
  printf("\nN=%d\n", N);

#ifdef LOG
  if(argc!=2){
    printf("Logging mode enabled. To run, specify logfile path as command line argument. Aborting...\n");
    exit(2);
  }
  FILE* logfile= fopen(argv[1], "a");
  fprintf(logfile, "%d;", N);
#endif

  run_cpu(g,N, groundTruth);
  run_GPUsimple(g,N, groundTruth);
  run_GPUblock(g,N, groundTruth);

#ifdef LOG
  fprintf(logfile, "\n");
  fclose(logfile);
#endif
  return 0;
}

