#include "hip/hip_runtime.h"
#include "utils.h"  // Define MAX_THRperBLK, MAX_THRperBLK_MULTI macros

//################# Kernels ###############
// GPUblock kernels
#define n MAX_THRperBLK2D
__global__ void phase1_krn(int* g){
  __shared__ int tile[n][n];
  // Load tile from global to shared
  tile[threadIdx.y][threadIdx.x]= g[ threadIdx.y*n + threadIdx.x ];
  __syncthreads();
  // Calculate APSP in tile
  for(int k=0; k<n; k++){
    if(tile[threadIdx.y][threadIdx.x] > tile[threadIdx.y][k]+tile[k][threadIdx.x])
      tile[threadIdx.y][threadIdx.x]= tile[threadIdx.y][k]+tile[k][threadIdx.x];
    __syncthreads();
  }
  // Save to global
  g[ threadIdx.y*n + threadIdx.x ]= tile[threadIdx.y][threadIdx.x];
}

__global__ void phase2_krn(int* g, const int* primaryTile, const int b, const int N){
  __shared__ int tile[n][n], primary[n][n];
  int blockIdx_xskip= (blockIdx.x >= b)? blockIdx.x+1: blockIdx.x;     // skip primary tile

  primary[threadIdx.y][threadIdx.x]= primaryTile[threadIdx.y*n + threadIdx.x];
  tile[threadIdx.y][threadIdx.x]= g[ blockIdx.y*N*n+ blockIdx_xskip*n + (threadIdx.y*N+threadIdx.x) ];
  __syncthreads();

  if(blockIdx.y)              // If column
    for(int k=0; k<n; k++){
      if(tile[threadIdx.y][threadIdx.x] > primary[k][threadIdx.x]+tile[threadIdx.y][k])
        tile[threadIdx.y][threadIdx.x]= primary[k][threadIdx.x]+tile[threadIdx.y][k];
      __syncthreads();
    }
  else                        // if row
    for(int k=0; k<n; k++){
      if(tile[threadIdx.y][threadIdx.x] > tile[k][threadIdx.x]+primary[threadIdx.y][k])
        tile[threadIdx.y][threadIdx.x]= tile[k][threadIdx.x]+primary[threadIdx.y][k];
      __syncthreads();
    }

  g[ blockIdx.y*N*n+ blockIdx_xskip*n + (threadIdx.y*N+threadIdx.x) ]= tile[threadIdx.y][threadIdx.x];
}

// Start: in blocks, not ints,relb= b-start 
__global__ void phase3_krn(int* g, const int* rowcol, const int b, const int N, const int xStart, const int yStart, const int rowL){
  __shared__ int tile[n][n], row[n][n], col[n][n];
  int blockIdx_xskip= (blockIdx.x >= b-xStart)? blockIdx.x+1: blockIdx.x;     // skip primary tile
  int blockIdx_yskip= (blockIdx.y >= b-yStart)? blockIdx.y+1: blockIdx.y;
  int x_t= blockIdx_xskip*n+threadIdx.x, y_t= blockIdx_yskip*n+threadIdx.y;     // tile coordinates
  row[threadIdx.y][threadIdx.x]=  rowcol[ n*xStart+ threadIdx.y*N + x_t ];
  col[threadIdx.y][threadIdx.x]=  rowcol[ n*N+ n*yStart+ blockIdx_yskip*n+ threadIdx.y*N + threadIdx.x ];
  tile[threadIdx.y][threadIdx.x]= g[ y_t*rowL + x_t ];
  __syncthreads();


  for(int k=0; k<n; k++){
    if(tile[threadIdx.y][threadIdx.x] > col[threadIdx.y][k]+row[k][threadIdx.x])
      tile[threadIdx.y][threadIdx.x]= col[threadIdx.y][k]+row[k][threadIdx.x];
    __syncthreads();
  }

  g[ y_t*rowL + x_t ]= tile[threadIdx.y][threadIdx.x];
}
#undef n

// TODO
// GPUblock_multiy kernels
#define n MAX_THRperBLK2D_MULTI
__global__ void phase1_multiy_krn(int* g, const int pstart, const int N){
  __shared__ int tile[n][n];
  tile[2*threadIdx.y][threadIdx.x]= g[ (pstart+2*threadIdx.y)*N + pstart+threadIdx.x ];
  tile[2*threadIdx.y+1][threadIdx.x]= g[ (pstart+2*threadIdx.y+1)*N + pstart+threadIdx.x ];
  __syncthreads();

  for(int k=0; k<n; k++){
    if(tile[2*threadIdx.y][threadIdx.x] > tile[2*threadIdx.y][k]+tile[k][threadIdx.x])
      tile[2*threadIdx.y][threadIdx.x]= tile[2*threadIdx.y][k]+tile[k][threadIdx.x];
    if(tile[2*threadIdx.y+1][threadIdx.x] > tile[2*threadIdx.y+1][k]+tile[k][threadIdx.x])
      tile[2*threadIdx.y+1][threadIdx.x]= tile[2*threadIdx.y+1][k]+tile[k][threadIdx.x];
    __syncthreads();
  }

  g[ (pstart+2*threadIdx.y)*N + pstart+threadIdx.x ]= tile[2*threadIdx.y][threadIdx.x];
  g[ (pstart+2*threadIdx.y+1)*N + pstart+threadIdx.x ]= tile[2*threadIdx.y+1][threadIdx.x];
}

__global__ void phase2_multiy_krn(int* g, const int pstart, const int primary_n, const int N){
  __shared__ int tile[n][n], primary[n][n];
  int blkIdx_skip= (blockIdx.x >= primary_n)? blockIdx.x+1: blockIdx.x;      // skip primary tile
  int x_t= (blockIdx.y)? blkIdx_skip*n+threadIdx.x: pstart+threadIdx.x;     // tile coordinates
  int y_t= (blockIdx.y)? pstart+2*threadIdx.y: blkIdx_skip*n+2*threadIdx.y;     // blkIdx,y? row: col
  int y_t1= (blockIdx.y)? pstart+2*threadIdx.y+1: blkIdx_skip*n+2*threadIdx.y+1;     // blkIdx,y? row: col
  primary[2*threadIdx.y][threadIdx.x]= g[ (pstart+2*threadIdx.y)*N + pstart+threadIdx.x ];
  primary[2*threadIdx.y+1][threadIdx.x]= g[ (pstart+2*threadIdx.y+1)*N + pstart+threadIdx.x ];
  tile[2*threadIdx.y][threadIdx.x]= g[y_t*N + x_t];
  tile[2*threadIdx.y+1][threadIdx.x]= g[y_t1*N + x_t];
  __syncthreads();
  
  if(blockIdx.y)
    for(int k=0; k<n; k++){
      if(tile[2*threadIdx.y][threadIdx.x] > primary[2*threadIdx.y][k]+tile[k][threadIdx.x])
        tile[2*threadIdx.y][threadIdx.x]= primary[2*threadIdx.y][k]+tile[k][threadIdx.x];
      if(tile[2*threadIdx.y+1][threadIdx.x] > primary[2*threadIdx.y+1][k]+tile[k][threadIdx.x])
        tile[2*threadIdx.y+1][threadIdx.x]= primary[2*threadIdx.y+1][k]+tile[k][threadIdx.x];
      __syncthreads();
    }
  else
    for(int k=0; k<n; k++){
      if(tile[2*threadIdx.y][threadIdx.x] > tile[2*threadIdx.y][k]+primary[k][threadIdx.x])
        tile[2*threadIdx.y][threadIdx.x]= tile[2*threadIdx.y][k]+primary[k][threadIdx.x];
      if(tile[2*threadIdx.y+1][threadIdx.x] > tile[2*threadIdx.y+1][k]+primary[k][threadIdx.x])
        tile[2*threadIdx.y+1][threadIdx.x]= tile[2*threadIdx.y+1][k]+primary[k][threadIdx.x];
      __syncthreads();
    }

  g[y_t*N + x_t]= tile[2*threadIdx.y][threadIdx.x];
  g[y_t1*N + x_t]= tile[2*threadIdx.y+1][threadIdx.x];
}

__global__ void phase3_multiy_krn(int* g, const int pstart, const int primary_n, const int N){
  __shared__ int tile[n][n], row[n][n], col[n][n];
  int blkIdx_xskip= (blockIdx.x >= primary_n)? blockIdx.x+1: blockIdx.x;     // skip primary tile
  int blkIdx_yskip= (blockIdx.y >= primary_n)? blockIdx.y+1: blockIdx.y;
  int x_t= blkIdx_xskip*n+threadIdx.x, y_t= blkIdx_yskip*n+2*threadIdx.y;     // tile coordinates
  int y_t1= blkIdx_yskip*n+2*threadIdx.y+1;     // tile coordinates
  row[2*threadIdx.y][threadIdx.x]= g[ (pstart+2*threadIdx.y)*N + x_t ];
  row[2*threadIdx.y+1][threadIdx.x]= g[ (pstart+2*threadIdx.y+1)*N + x_t ];
  col[2*threadIdx.y][threadIdx.x]= g[ y_t*N + pstart+threadIdx.x   ];
  col[2*threadIdx.y+1][threadIdx.x]= g[ y_t1*N + pstart+threadIdx.x   ];
  tile[2*threadIdx.y][threadIdx.x]= g[y_t*N + x_t];
  tile[2*threadIdx.y+1][threadIdx.x]= g[y_t1*N + x_t];
  __syncthreads();

  for(int k=0; k<n; k++){
    if(tile[2*threadIdx.y][threadIdx.x] > col[2*threadIdx.y][k]+row[k][threadIdx.x])
      tile[2*threadIdx.y][threadIdx.x]= col[2*threadIdx.y][k]+row[k][threadIdx.x];
    if(tile[2*threadIdx.y+1][threadIdx.x] > col[2*threadIdx.y+1][k]+row[k][threadIdx.x])
      tile[2*threadIdx.y+1][threadIdx.x]= col[2*threadIdx.y+1][k]+row[k][threadIdx.x];
    __syncthreads();
  }

  g[y_t*N + x_t]= tile[2*threadIdx.y][threadIdx.x];
  g[y_t1*N + x_t]= tile[2*threadIdx.y+1][threadIdx.x];
}
#undef n

//############# Kernel Wrappers ############
void phase1(const dim3 gs, const dim3 bs, int* g){
  phase1_krn<<<gs,bs>>>(g);
}
void phase2(const dim3 gs, const dim3 bs, int* g, const int* primaryTile, const int b, const int N){
  phase2_krn<<<gs,bs>>>(g,primaryTile,b,N);
}
void phase3(const dim3 gs, const dim3 bs, int* g, const int* rowcol, const int b, const int N, const int xStart, const int yStart, const int rowL){
  phase3_krn<<<gs,bs>>>(g,rowcol, b,N,xStart,yStart,rowL);
}

void phase1_multiy(const dim3 gs, const dim3 bs, int* g, const int pstart, const int N){
  phase1_multiy_krn<<<gs,bs>>>(g,pstart,N);
}
void phase2_multiy(const dim3 gs, const dim3 bs, int* g, const int pstart, const int primary_n, const int N){
  phase2_multiy_krn<<<gs,bs>>>(g,pstart,primary_n,N);
}
void phase3_multiy(const dim3 gs, const dim3 bs, int* g, const int pstart, const int primary_n, const int N){
  phase3_multiy_krn<<<gs,bs>>>(g,pstart,primary_n,N);
}

