#include "DPtr.h"

#define gpuErrchk(ans) gpuAssert((ans), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

DPtr::DPtr(int N) { gpuErrchk(hipMalloc(&data_, N*sizeof(int))); }
DPtr::~DPtr() { hipFree(data_); }
void DPtr::copy(int* a, const int N, const Dir dir, const devOffset=0) {
  if(dir == Dir::H2D) gpuErrchk(hipMemcpy(data_+devOffset, a, sizeof(int)*N, hipMemcpyHostToDevice));
  else gpuErrchk(hipMemcpy(a, data_, sizeof(int)*N, hipMemcpyDeviceToHost));
}

int* DPtr::get() const { return data_; }
DPtr::operator int*() const { return data_; }

