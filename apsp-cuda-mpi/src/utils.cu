#include "utils.h"
#include "DPtr.h"
#include <hip/hip_runtime.h>

template<>
HPinPtr<int>::HPinPtr(const int N) {
  gpuErrchk(hipHostAlloc(&data_, N*sizeof(int), hipHostMallocDefault));
}
template<>
HPinPtr<int>::~HPinPtr() { hipHostFree(data_); }
