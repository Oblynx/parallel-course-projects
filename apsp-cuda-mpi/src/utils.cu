#include "utils.h"
#include "DPtr.h"
#include <hip/hip_runtime.h>

HPinPtr::HPinPtr(const int N) {
  gpuErrchk(hipHostAlloc(&data_, N*sizeof(int), hipHostMallocDefault));
}
HPinPtr::~HPinPtr() { hipHostFree(data_); }
