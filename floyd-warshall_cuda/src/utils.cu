#include "utils.h"
#include "DPtr.cuh"
#include <hip/hip_runtime.h>

template<>
HPinPtr<int>::HPinPtr(const int N) { gpuErrchk(hipHostAlloc(&data_, N*sizeof(int), hipHostMallocDefault)); }
template<>
HPinPtr<int>::~HPinPtr() { hipHostFree(data_); }
